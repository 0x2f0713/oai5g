#include "hip/hip_runtime.h"
/*
 * Licensed to the OpenAirInterface (OAI) Software Alliance under one or more
 * contributor license agreements.  See the NOTICE file distributed with
 * this work for additional information regarding copyright ownership.
 * The OpenAirInterface Software Alliance licenses this file to You under
 * the OAI Public License, Version 1.1  (the "License"); you may not use this file
 * except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *      http://www.openairinterface.org/?page_id=698
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *-------------------------------------------------------------------------------
 * For more information about the OpenAirInterface (OAI) Software Alliance:
 *      contact@openairinterface.org
 */

 /*! \file CUDA_phy_procedure.cu
 * \brief Create and Implementation of beamforming and ifft in gpu
 * \author TY Hsu, CW Chang
 * \date 2018
 * \version 0.1
 * \company ISIP@NCTU and Eurecom
 * \email: tyhsu@cs.nctu.edu.tw, zhang0756107.cs07g@nctu.edu.tw
 * \note
 * \warning
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "CUDA/checkError.h"
#include "CUDA/struct.h"
#include "CUDA/cuda_struct.h"

__global__ void cu_intToComplex(int *input, Complex *signal){
	int id = blockIdx.x*1024 + threadIdx.x;
	signal[id].x = ((short*)&input[id])[0];
	signal[id].y = ((short*)&input[id])[1];
}

__global__ void cu_ComplexToInt(int *output, Complex *signal){
	int id = blockIdx.x*1024 + threadIdx.x;
	((short*)&output[id])[0] = round(signal[id].x);
	((short*)&output[id])[1] = round(signal[id].y);
}

__global__ void cu_CP_fft_resultin(unsigned char nb_prefix_samples, int *input, int *output, int fftsize, int nb_symbols){
	int id = blockIdx.x*1024 + threadIdx.x;
	int elementId = id%fftsize;
	int symbolId = id/fftsize;
	int slotId = symbolId/nb_symbols;
	int symbIdinSlot = symbolId%nb_symbols;
	int slotElmtNum = fftsize*(nb_symbols+1);
	int CPElmtNum = fftsize+nb_prefix_samples;

	int offset = slotId*slotElmtNum + symbIdinSlot*CPElmtNum;
	output[offset + nb_prefix_samples + elementId] = input[id];
	if(elementId >= fftsize-nb_prefix_samples){
		output[offset + (fftsize-nb_prefix_samples)] = input[id];
	}
}

__global__ void cu_CP0_fft_resultin(unsigned char nb_prefix_samples0, unsigned char nb_prefix_samples, 
								int *input, int *output, int fftsize, int nb_symbols){
	int id = blockIdx.x*1024 + threadIdx.x;
	int elementId = id%fftsize;
	int symbolId = id/fftsize;
	int slotId = symbolId/nb_symbols;
	int symbIdinSlot = symbolId%nb_symbols;
	int slotElmtNum = fftsize*(nb_symbols+1);
	int CP0ElmtNum = fftsize+nb_prefix_samples0;
	int CPElmtNum = fftsize+nb_prefix_samples;

	if(symbIdinSlot==0){
		int offset = slotId*slotElmtNum;
		output[offset + nb_prefix_samples0+ elementId] = input[id];
		if(elementId >= fftsize-nb_prefix_samples0){
			output[offset + (fftsize-nb_prefix_samples0)] = input[id];
		} 
	}else{
		int offset = slotId*slotElmtNum + CP0ElmtNum + (symbIdinSlot-1)*CPElmtNum;
		output[offset + nb_prefix_samples + elementId] = input[id];
		if(elementId >= fftsize-nb_prefix_samples){
			output[offset + (fftsize-nb_prefix_samples)] = input[id];
		}
	}
}

extern "C" void CUDA_ifft_ofdm( int **output, 
				int fftsize, 
				unsigned char nb_symbols, 
				unsigned char nb_prefix_samples,
				unsigned char nb_prefix_samples0,
				int nb_tx,
				int Ncp,
				Extension_t etype){
	//for(int i=0; i<fftsize; i++) printf("%d+%di\n", ((short*)&input[0][i])[0], ((short*)&input[0][i])[1]);
	
	int *d_txdataF_BF = cu_ru.d_txdataF_BF;
	int *d_data_wCP = cu_ru.d_data_wCP;
	Complex *d_signal = cu_ru.d_signal;
	hipfftHandle plan = cu_ru.plan; 

	/*
	for(int aa=0; aa<nb_tx; aa++){
		int elementNum = fftsize*nb_symbols;
		gpuErrchk( hipMemcpy(&d_data[aa*elementNum], input[aa], sizeof(int)*elementNum, hipMemcpyHostToDevice) );
	}*/

	int threadNum = 1024;
	int blockNum = fftsize*nb_symbols*nb_tx / threadNum;
	cu_intToComplex<<<blockNum, threadNum>>>(d_txdataF_BF, d_signal);
	//CHECK_STATE("cu_intToComplex");

	cufftErrchk( hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

	cu_ComplexToInt<<<blockNum, threadNum>>>(d_txdataF_BF, d_signal);
	//CHECK_STATE("cu_ComplexToInt");


	//only do cyclic_prefix, suffix/none not finish yet.
	if(Ncp==1){
		cu_CP_fft_resultin<<<blockNum, threadNum>>>(nb_prefix_samples, d_txdataF_BF, d_data_wCP, fftsize, nb_symbols);
		//CHECK_STATE("cu_CP_fft_resultin");	
	}else{
		cu_CP0_fft_resultin<<<blockNum, threadNum>>>(nb_prefix_samples0, nb_prefix_samples, d_txdataF_BF, d_data_wCP, fftsize, nb_symbols);
		//CHECK_STATE("cu_CP0_fft_resultin");	
	}

	//write back gpu->cpu
	for(int aa=0; aa<nb_tx; aa++){
		gpuErrchk( hipMemcpy(output[aa], &d_data_wCP[aa*(nb_symbols+1)*fftsize], fftsize*(nb_symbols+1)*sizeof(int), hipMemcpyDeviceToHost) );
	}
	hipDeviceSynchronize();
	

}

__global__ void conjMul(int *d_x1, int *d_x2, int *d_y, int aa, int div, int fftsize, int nb_symbols){
	int id = blockIdx.x*1024 + threadIdx.x;
	int *x1 = &d_x1[id];
	int *x2 = &d_x2[id%fftsize];
	int *y = &d_y[aa*fftsize*nb_symbols + id];

	int re, im;
	//conj(x1) * x2
	re = ((short*)x1)[0]*((short*)x2)[0] + ((short*)x1)[1]*((short*)x2)[1];
	im = ((short*)x1)[0]*((short*)x2)[1] - ((short*)x1)[1]*((short*)x2)[0];

	re = re / div;
	im = im / div;

	((short*)y)[0] += re; 
	((short*)y)[1] += im;	

}


extern "C" void CUDA_beam_precoding(int **txdataF, int ***weight, int L_ssb, int shift, int fftsize, int nb_symbols, int nb_antenna_ports, int nb_tx){

	//initial BF data;
	gpuErrchk( hipMemset(cu_ru.d_txdataF_BF, 0, fftsize*nb_symbols*sizeof(int)*nb_tx) );
	//move data to gpu
	for(int p=0; p<nb_antenna_ports; p++){
		gpuErrchk( hipMemcpy(cu_ru.d_txdataF[p], txdataF[p], fftsize*sizeof(int)*nb_symbols, hipMemcpyHostToDevice) );	
	}

	
	int threadNum = 1024;
	int blockNum = fftsize*nb_symbols/threadNum;
	int div = 1<<shift;
	for(int aa=0; aa<nb_tx; aa++){
		for(int p=0; p<nb_antenna_ports; p++){
			if((L_ssb>>p) & 0x01){
				gpuErrchk( hipMemcpy(cu_ru.d_weight[p][aa], weight[p][aa], fftsize*sizeof(int), hipMemcpyHostToDevice) );
				conjMul<<<blockNum, threadNum>>>(cu_ru.d_txdataF[p], cu_ru.d_weight[p][aa], 
							cu_ru.d_txdataF_BF, aa, div, fftsize, nb_symbols);
			}
		}
	}


}

