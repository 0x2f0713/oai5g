#include "hip/hip_runtime.h"
/*! \file PHY\CUDA/LTE_TRANSPORT/turbo_rx_gpu.cu
* \brief turbo decoder using gpu 
* \author TerngYin Hsu, JianYa Chu
* \date 2018
* \version 0.1
* \company ISIP LAB/NCTU CS  
* \email: tyhsu@cs.nctu.edu.tw
* \note
* \warning
*/

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>


#include "turbo_parm.h"
#include "PHY/CODING/extern_3GPPinterleaver.h"
//#include "extern_interleaver.h"
#include "PHY/CODING/defs.h"
#include "turbo_rx_gpu.h"
//#include "crc_byte.h"
//#include "turbo_rx.h"
//#include "PHY/CODING/extern_table_gpu.h"
//#include "extern_table_gpu.h"
#include "PHY/defs.h"
//typedef int16_t llr_t;

#define CRC24_A 0
#define CRC24_B 1
#define CRC16 2
#define CRC8 3

int intable_h[188][6144];
int detable_h[188][6144];

void free_ptr()
{
    hipFree(turbo_parm->sys_d);
	hipFree(turbo_parm->sys1_d);
	hipFree(turbo_parm->sys2_d);
	hipFree(turbo_parm->ypar1_d);
	hipFree(turbo_parm->ypar2_d);
	hipFree(turbo_parm->alpha_d);
	hipFree(turbo_parm->alpha_pre_1);
	hipFree(turbo_parm->alpha_pre_2);
	hipFree(turbo_parm->beta_pre_1);
	hipFree(turbo_parm->beta_pre_2);
	hipFree(turbo_parm->ext_d);
	hipFree(turbo_parm->ext2_d);
	hipFree(turbo_parm->decode_ext2);
	hipFree(turbo_parm->decode_tmp);
	hipHostFree(turbo_parm->decode_h);
	free(turbo_parm);

	char i;
	for(i=0;i<2;i++)
	{
		hipStreamDestroy(cuda_parm.stream[i]);
	}
}

__constant__ int alpha_table_0[32];
__constant__ int alpha_table_1[32];
__constant__ int beta_table_0[32];
__constant__ int beta_table_1[32];
__constant__ float alpha_par_table_0[32];
__constant__ float alpha_par_table_1[32];
__constant__ float beta_par_table_0[32];
__constant__ float beta_par_table_1[32];
__constant__ int interleaver[6144];
__constant__ int de_interleaver[6144];

void init_alloc()
{
	size_t pitch;
	
    hipDeviceProp_t deviceprop;
    hipGetDeviceProperties(&deviceprop,0);

    hipSetDeviceFlags(hipDeviceMapHost);

    if(deviceprop.canMapHostMemory!=1)
        printf("hipError_t:cannot map host to device memory\n");

    hipError_t result;
	
	turbo_parm = (turbo_parm_s*)malloc(sizeof(turbo_parm_s));
	
	// allocate CUDA memory 
	result = hipMallocPitch((void**)&turbo_parm->sys_d, &pitch, 16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->sys_d failed, err_num=%d\n",result);
	
	result = hipMallocPitch((void**)&turbo_parm->sys1_d, &pitch, 16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->sys1_d failed, err_num=%d\n",result);
	
	result = hipMallocPitch((void**)&turbo_parm->sys2_d, &pitch, 16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->sys2_d failed, err_num=%d\n",result);
	
    result = hipMallocPitch((void**)&turbo_parm->ypar1_d, &pitch, 16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->ypar1_d failed, err_num=%d\n",result);
	
	result = hipMallocPitch((void**)&turbo_parm->ypar2_d, &pitch, 16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->ypar2_d failed, err_num=%d\n",result);
	
	result =  hipMallocPitch((void**)&turbo_parm->alpha_d, &pitch, 16*8*(6144+648)*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->alpha failed, err_num=%d\n",result);
	
	result =  hipMallocPitch((void**)&turbo_parm->alpha_pre_1, &pitch, 16*32*162*4*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->alpha_pre_1 failed, err_num=%d\n",result);
	
	result =  hipMallocPitch((void**)&turbo_parm->alpha_pre_2, &pitch, 16*32*162*4*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->alpha_pre_2 failed, err_num=%d\n",result);
	
	result = hipMallocPitch((void**)&turbo_parm->beta_pre_1, &pitch, 16*32*162*4*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->beta_pre1 failed, err_num=%d\n",result);
	
	result = hipMallocPitch((void**)&turbo_parm->beta_pre_2, &pitch, 16*32*162*4*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->beta_pre2 failed, err_num=%d\n",result);
	
	result =  hipMallocPitch((void**)&turbo_parm->ext_d, &pitch, 16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->ext_d failed, err_num=%d\n",result);
	
	result = hipMallocPitch((void**)&turbo_parm->ext2_d, &pitch, 16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->ext2_d failed, err_num=%d\n",result);

	result = hipMallocPitch((void**)&turbo_parm->decode_ext2, &pitch, 3*16*6144*sizeof(llr_t) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->ext2_d failed, err_num=%d\n",result);

	result = hipMallocPitch((void**)&turbo_parm->decode_tmp, &pitch, 3*16*6144*sizeof(int) ,1);
	if(result!=hipSuccess)
		printf("hipMalloc turbo_parm->ext2_d failed, err_num=%d\n",result);

	result = hipHostAlloc((void**)&turbo_parm->decode_h,3*16*768*sizeof(unsigned char), hipHostMallocMapped);
	if(result!=hipSuccess)
		printf("hipHostAlloc turbo_parm->decode_h filaed, err_num=%d\n",result);
	
	// get device pointer
	result = hipHostGetDevicePointer(&turbo_parm->decode_d, turbo_parm->decode_h, 0);
	if(result!=hipSuccess)
		printf("cuda get device pinter decode_d failed, err_num=%d\n",result);

	// memset for mem
	hipMemset(turbo_parm->ext2_d,0,16*6144*sizeof(llr_t));
	hipMemset(turbo_parm->alpha_d,0,16*8*(6144+648)*sizeof(llr_t));
	hipMemset(turbo_parm->decode_tmp,0,16*6144*sizeof(llr_t));
	//memset(turbo_parm->decode_h,0,16*768*sizeof(char));

	// init table for decoder
	int a_table_0[32]={0,3,4,7,1,2,5,6,8,11,12,15,9,10,13,14,16,19,20,23,17,18,21,22,24,27,28,31,25,26,29,30};
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_table_0),a_table_0,32*sizeof(int));
	int a_table_1[32]={1,2,5,6,0,3,4,7,9,10,13,14,8,11,12,15,17,18,21,22,16,19,20,23,25,26,29,30,24,27,28,31};
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_table_1),a_table_1,32*sizeof(int));
	
	float a_p_table_0[32] = {0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0};
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_par_table_0),a_p_table_0,32*sizeof(llr_t));
	float a_p_table_1[32] = {1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0 ,1.0 ,0.0 ,0.0 ,1.0};
	hipMemcpyToSymbol(HIP_SYMBOL(alpha_par_table_1),a_p_table_1,32*sizeof(llr_t));
	
	int b_table_0[32]={0,4,5,1,2,6,7,3,8,12,13,9,10,14,15,11,16,20,21,17,18,22,23,19,24,28,29,25,26,30,31,27};
	hipMemcpyToSymbol(HIP_SYMBOL(beta_table_0),b_table_0,32*sizeof(int));
	int b_table_1[32]={4,0,1,5,6,2,3,7,12,8,9,13,14,10,11,15,20,16,17,21,22,18,19,23,28,24,25,29,30,26,27,31};
	hipMemcpyToSymbol(HIP_SYMBOL(beta_table_1),b_table_1,32*sizeof(int));
	
	float b_p_table_0[32] = {0.0 ,0.0 ,1.0 ,1.0 ,1.0 ,1.0 ,0.0 ,0.0 , 0.0 ,0.0 ,1.0 ,1.0 ,1.0 ,1.0 ,0.0 ,0.0 , 0.0 ,0.0 ,1.0 ,1.0 ,1.0 ,1.0 ,0.0 ,0.0 , 0.0 ,0.0 ,1.0 ,1.0 ,1.0 ,1.0 ,0.0 ,0.0};
	hipMemcpyToSymbol(HIP_SYMBOL(beta_par_table_0),b_p_table_0,32*sizeof(llr_t));
	float b_p_table_1[32] = {1.0 ,1.0 ,0.0 ,0.0 ,0.0 ,0.0 ,1.0 ,1.0 , 1.0 ,1.0 ,0.0 ,0.0 ,0.0 ,0.0 ,1.0 ,1.0 , 1.0 ,1.0 ,0.0 ,0.0 ,0.0 ,0.0 ,1.0 ,1.0 , 1.0 ,1.0 ,0.0 ,0.0 ,0.0 ,0.0 ,1.0 ,1.0};
	hipMemcpyToSymbol(HIP_SYMBOL(beta_par_table_1),b_p_table_1,32*sizeof(llr_t));
	
	
	// build de-interleaver table and interleaver table
	int i, j;
	unsigned long n;
	unsigned short f1, f2;
	for(j=0;j<188;j++)
	{
		n = f1f2mat[j].nb_bits;
		f1 = f1f2mat[j].f1;
		f2 = f1f2mat[j].f2;
		for(i=0;i<n;i++)
		{
			intable_h[j][i] = (((f1+f2*i)%n)*i)%n;
			detable_h[j][(((f1+f2*i)%n)*i)%n] = i;
		}
	}

	// for crc check and stream create
	for(i=0;i<2;i++)
	{
		hipStreamCreate(&cuda_parm.stream[i]);
	}

	for(i=0;i<3;i++)
	{
		hipEventCreate(&cuda_parm.s_check[i]);
	}
}
__device__ void compute_alpha(float* sys, float* sys1, float* sys2,
							  float* par, 
						      float* alpha, float* alpha_tmp,
						      float* alpha_pre_1, float* alpha_pre_2,
						      int num_per_block, int iteration_cnt, int decoder_id, int n, int codeword_num)
{
	int alpha_start = blockIdx.y*(n+gridDim.x*4)*8 + blockIdx.x*(num_per_block+1)*8*4;
	int index = blockIdx.y*n + blockIdx.x*num_per_block*4 + num_per_block*threadIdx.y;
	llr_t r0, r1;
	char i;
	
	alpha_tmp[threadIdx.x + 8*threadIdx.y] = 0;
	if(!(iteration_cnt==0 || (iteration_cnt==1 && decoder_id==2)))
	{
		if(!(blockIdx.x==0 && threadIdx.y==0))
		{
			if(decoder_id==1)
				alpha_tmp[threadIdx.x + 8*threadIdx.y] = alpha_pre_1[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y -8];
			else
				alpha_tmp[threadIdx.x + 8*threadIdx.y] = alpha_pre_2[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y -8];
		}
	}
	alpha[alpha_start + threadIdx.x + 8*threadIdx.y] = alpha_tmp[threadIdx.x + 8*threadIdx.y];
	__syncthreads();
	
	for(i=0;i<num_per_block;i++)
	{
		if(decoder_id==1)
		{
			r0 = alpha_par_table_0[threadIdx.x + 8*threadIdx.y]*par[index + i];
			r1 = sys1[index + i] + alpha_par_table_1[threadIdx.x + 8*threadIdx.y]*par[index + i];
		}
		else
		{
			r0 = alpha_par_table_0[threadIdx.x + 8*threadIdx.y]*par[index + i];
			r1 = sys2[index + i] + alpha_par_table_1[threadIdx.x + 8*threadIdx.y]*par[index + i];
		}
		alpha[alpha_start + (i+1)*32 + threadIdx.x + 8*threadIdx.y] = fmaxf(alpha_tmp[alpha_table_0[threadIdx.x+8*threadIdx.y]] + r0, alpha_tmp[alpha_table_1[threadIdx.x + 8*threadIdx.y]] + r1);
		__syncthreads();
		alpha_tmp[threadIdx.x + 8*threadIdx.y] = alpha[alpha_start + (i+1)*32 + threadIdx.x + 8*threadIdx.y];
		if(i==num_per_block-1)
		{
			if(decoder_id==1)
				alpha_pre_1[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y] = alpha_tmp[threadIdx.x + 8*threadIdx.y];
			else
				alpha_pre_2[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y] = alpha_tmp[threadIdx.x + 8*threadIdx.y];
		}
	}
	
}

__device__ void compute_beta_ext(float* sys, float* sys1, float* sys2,
							float* par, 
							float* alpha, float* beta_now, float* beta_next, 
							float* beta_pre_1, float* beta_pre_2,
							float* ext_tmp0, float* ext_tmp1,
							float* ext, float* ext2, float* decode_ext2,
							int num_per_block, int iteration_cnt, int decoder_id, int n, int codeword_num)
{
	llr_t a, r0, r1, max_0, max_1;
	int alpha_start = blockIdx.y*(n+gridDim.x*4)*8 + blockIdx.x*(num_per_block+1)*8*4;
	int index = blockIdx.y*n + blockIdx.x*num_per_block*4 + threadIdx.y*num_per_block;
	int index2;
	char i,j;
	
	beta_now[threadIdx.x + 8*threadIdx.y] = 0;
	if(!(iteration_cnt==0 || (iteration_cnt==1 && decoder_id==2)))
	{
		if(!(blockIdx.x==gridDim.x-1 && threadIdx.y==3))
		{
			if(decoder_id==1)
				beta_now[threadIdx.x + 8*threadIdx.y] = beta_pre_1[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y +8];
			else
				beta_now[threadIdx.x + 8*threadIdx.y] = beta_pre_2[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y +8];
		}
	}
	__syncthreads();
	
	for(i=num_per_block-1;i>=0;i--)
	{
		if(decoder_id==1)
		{
			r0 = beta_par_table_0[threadIdx.x + 8*threadIdx.y]*par[index + i];
			r1 = sys1[index + i] + beta_par_table_1[threadIdx.x + 8*threadIdx.y]*par[index + i];
		}
		else
		{
			r0 = beta_par_table_0[threadIdx.x + 8*threadIdx.y]*par[index + i];
			r1 = sys2[index + i] + beta_par_table_1[threadIdx.x + 8*threadIdx.y]*par[index + i];
		}
		a = alpha[alpha_start + 32*i + threadIdx.x + 8*threadIdx.y];
		beta_next[threadIdx.x + 8*threadIdx.y] = fmaxf(beta_now[beta_table_0[threadIdx.x+8*threadIdx.y]] + r0, beta_now[beta_table_1[threadIdx.x + 8*threadIdx.y]] + r1);
		if(i==0)
		{
			if(decoder_id==1)
				beta_pre_1[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y] = beta_next[threadIdx.x + 8*threadIdx.y];
			else
				beta_pre_2[blockIdx.y*gridDim.x*32 + blockIdx.x*32 + threadIdx.x + 8*threadIdx.y] = beta_next[threadIdx.x + 8*threadIdx.y];
		}
		ext_tmp0[((num_per_block-1-i)&7)*32 + threadIdx.x + 8*threadIdx.y] = a + r0 + beta_now[beta_table_0[threadIdx.x + 8*threadIdx.y]];
		ext_tmp1[((num_per_block-1-i)&7)*32 + threadIdx.x + 8*threadIdx.y] = a + r1 + beta_now[beta_table_1[threadIdx.x + 8*threadIdx.y]];
		__syncthreads();
		beta_now[threadIdx.x + 8*threadIdx.y] = beta_next[threadIdx.x + 8*threadIdx.y];
		
		if(((num_per_block-1-i)&7)==7)
		{
			max_0 = ext_tmp0[(7-threadIdx.x)*32 + threadIdx.x + 8*threadIdx.y];
			max_1 = ext_tmp1[(7-threadIdx.x)*32 + threadIdx.x + 8*threadIdx.y];
			for(j=1;j<8;j++)
			{
				index2 = (threadIdx.x + j)&7;
				max_0 = fmaxf(max_0, ext_tmp0[(7-threadIdx.x)*32 + index2 + 8*threadIdx.y]);
				max_1 = fmaxf(max_1, ext_tmp1[(7-threadIdx.x)*32 + index2 + 8*threadIdx.y]);
			}
			index2 = blockIdx.x*num_per_block*4 + threadIdx.y*num_per_block + i + threadIdx.x;
			if(decoder_id==1)
			{
				ext[blockIdx.y*n + index2] = max_1 - max_0 - sys1[blockIdx.y*n + index2] + sys[blockIdx.y*n + index2];
				sys2[blockIdx.y*n + de_interleaver[index2]] = ext[blockIdx.y*n + index2];
			}
			else
			{
				ext2[blockIdx.y*n + index2] = max_1 - max_0;
				if(iteration_cnt >= 3)
				{
					decode_ext2[(iteration_cnt-3)*gridDim.y*n + blockIdx.y*n + interleaver[index2]] = max_1 - max_0;
				}
				sys1[blockIdx.y*n + interleaver[index2]] = max_1 - max_0 - ext[blockIdx.y*n + interleaver[index2]] + sys[blockIdx.y*n + interleaver[index2]];
			}
		}
		else if(i==0 && (num_per_block&7)!=0 && threadIdx.x < (num_per_block&7))
		{
			max_0 = ext_tmp0[((num_per_block&7)-1-threadIdx.x)*32 + threadIdx.x + 8*threadIdx.y];
			max_1 = ext_tmp1[((num_per_block&7)-1-threadIdx.x)*32 + threadIdx.x + 8*threadIdx.y];
			for(j=1;j<8;j++)
			{
				index2 = (threadIdx.x + j)&7;
				max_0 = fmaxf(max_0, ext_tmp0[((num_per_block&7)-1-threadIdx.x)*32 + index2 + 8*threadIdx.y]);
				max_1 = fmaxf(max_1, ext_tmp1[((num_per_block&7)-1-threadIdx.x)*32 + index2 + 8*threadIdx.y]);
			}
			index2 = blockIdx.x*num_per_block*4 + threadIdx.y*num_per_block + i + threadIdx.x;
			if(decoder_id==1)
			{
				ext[blockIdx.y*n + index2] = max_1 - max_0 - sys1[blockIdx.y*n + index2] + sys[blockIdx.y*n + index2];
				sys2[blockIdx.y*n + de_interleaver[index2]] = ext[blockIdx.y*n + index2];
			}
			else
			{
				ext2[blockIdx.y*n + index2] = max_1 - max_0;
				if(iteration_cnt >= 3)
				{
					decode_ext2[(iteration_cnt-3)*gridDim.y*n + blockIdx.y*n + interleaver[index2]] = max_1 - max_0;
				}
				sys1[blockIdx.y*n + interleaver[index2]] = max_1 - max_0 - ext[blockIdx.y*n + interleaver[index2]] + sys[blockIdx.y*n + interleaver[index2]];
			}
		}
	}
}

__global__ void log(float* sys, float* sys1, float* sys2,
					float* par,
					float* alpha, 
					float* alpha_pre_1, float* alpha_pre_2, float* beta_pre_1, float* beta_pre_2,
					float* ext, float* ext2, float* decode_ext2,
					int num_per_block, int iteration_cnt, int decoder_id, int n, int codeword_num
					)
{
	__shared__ llr_t alpha_tmp[32];
	__shared__ llr_t beta_tmp[32];
	__shared__ llr_t ext_tmp0[32*8];
	__shared__ llr_t ext_tmp1[32*8];
	
	compute_alpha(sys, sys1, sys2,
				   par,
				   alpha, alpha_tmp,
				   alpha_pre_1, alpha_pre_2,
				   num_per_block, iteration_cnt, decoder_id, n , codeword_num
				   );
	__syncthreads();			  
	
	compute_beta_ext(sys, sys1, sys2,
					 par,
					 alpha, alpha_tmp,beta_tmp,
					 beta_pre_1, beta_pre_2,
					 ext_tmp0, ext_tmp1,
					 ext, ext2, decode_ext2,
					 num_per_block, iteration_cnt, decoder_id, n, codeword_num
					);
		 
}

// for decoding
__global__ void decode(llr_t* decode_ext2, unsigned char* decode_d, int* decode_tmp, int n, int decode_len, int iteration_cnt)
{
	int i, j;
	for(i=threadIdx.x; i<n; i+=256)
	{
		decode_tmp[iteration_cnt*gridDim.x*n + blockIdx.x*n + i] = 0;
		if(decode_ext2[iteration_cnt*gridDim.x*n + blockIdx.x*n + i] > 0)
		{
			decode_tmp[iteration_cnt*gridDim.x*n + blockIdx.x*n + i] = 1 << (7-(i&7));
		}
	}
	__syncthreads();
	for(i=threadIdx.x; i<decode_len; i+=256)
	{
		decode_d[iteration_cnt*gridDim.x*decode_len + blockIdx.x*decode_len + i] = 0;
		for(j=0;j<8;j++)
		{
			decode_d[iteration_cnt*gridDim.x*decode_len + blockIdx.x*decode_len + i] += decode_tmp[iteration_cnt*gridDim.x*n + blockIdx.x*n + i*8 + j];
		}
	}
}

//#define TIME_EST

unsigned char phy_threegpplte_turbo_decoder_gpu(short **y,
        unsigned char **decoded_bytes,
		unsigned int codeword_num,
        unsigned short n,
        unsigned short f1,
        unsigned short f2,
        unsigned char max_iterations,
        unsigned char crc_type,
        unsigned char *f_tmp,
		unsigned char* ret)
{
	unsigned int i,j,iind,k;
#ifdef TIME_EST
	hipEventCreate(&cuda_parm.e_start);
	hipEventCreate(&cuda_parm.e_stop);
	cuda_parm.e_time = 0;
	hipEventRecord(cuda_parm.e_start, cuda_parm.stream[0]);
#endif

    llr_t sys_h[n*codeword_num], ypar1_h[n*codeword_num], ypar2_h[n*codeword_num];
	
	unsigned char iteration_cnt=0;
	unsigned int crc,oldcrc,crc_len;
	uint8_t temp;
	unsigned char F;
	
	if (crc_type > 3) {
		printf("Illegal crc length!\n");
		return 255;
	}
	
	for (iind=0; f1f2mat[iind].nb_bits!=n && iind <188; iind++);
	
	if ( iind == 188 ) {
		printf("Illegal frame length!\n");
		return 255;
	}
	
	switch (crc_type) {
		case CRC24_A:
		case CRC24_B:
			crc_len=3;
			break;

		case CRC16:
			crc_len=2;
			break;

		case CRC8:
			crc_len=1;
			break;

		default:
			crc_len=3;
	}
	
	// fetch data for each codeword
#ifdef TIME_EST
	hipEventCreate(&cuda_parm.f_start);
	hipEventCreate(&cuda_parm.f_stop);
	cuda_parm.f_time = 0;
	hipEventRecord(cuda_parm.f_start,cuda_parm.stream[0]);
#endif
	short* yp;
	for(i=0;i<codeword_num;i++)
	{
		yp = y[i];
		for(j=0;j<n;j++)
		{
			sys_h[j+n*i] = *yp;
			ypar1_h[j+n*i] = *(yp+1);
			ypar2_h[j+n*i] = *(yp+2);
			yp+=3;
		}
	}
#ifdef TIME_EST
	hipEventRecord(cuda_parm.f_stop,cuda_parm.stream[0]);
	hipEventSynchronize(cuda_parm.f_stop);
	cuda_parm.f_time = 0;
	hipEventElapsedTime(&cuda_parm.f_time, cuda_parm.f_start, cuda_parm.f_stop);
	printf("Fetech data time = %f ms\n",cuda_parm.f_time);
#endif

	// for kernel  memcpy
#ifdef TIME_EST
	hipEventCreate(&cuda_parm.m_start);
	hipEventCreate(&cuda_parm.m_stop);
	hipEventRecord(cuda_parm.m_start, cuda_parm.stream[0]);
#endif
	hipMemcpyAsync(turbo_parm->sys_d,sys_h,codeword_num*n*sizeof(llr_t),hipMemcpyHostToDevice, cuda_parm.stream[0]);
	hipMemcpyAsync(turbo_parm->ypar1_d,ypar1_h,codeword_num*n*sizeof(llr_t),hipMemcpyHostToDevice, cuda_parm.stream[0]);
	hipMemcpyAsync(turbo_parm->ypar2_d,ypar2_h,codeword_num*n*sizeof(llr_t),hipMemcpyHostToDevice, cuda_parm.stream[0]);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(interleaver), intable_h[iind],n*sizeof(int), 0, hipMemcpyHostToDevice,  cuda_parm.stream[0]);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(de_interleaver), detable_h[iind],n*sizeof(int), 0, hipMemcpyHostToDevice,  cuda_parm.stream[0]);
#ifdef TIME_EST
	hipEventRecord(cuda_parm.m_stop, cuda_parm.stream[0]);
	hipEventSynchronize(cuda_parm.m_stop);	
	cuda_parm.m_time=0;
	hipEventElapsedTime(&cuda_parm.m_time, cuda_parm.m_start, cuda_parm.m_stop);
	printf("Memcpy Time For Kernel = %f ms\n",cuda_parm.m_time);
#endif

#ifdef TIME_EST	
	cuda_parm.t_time = 0;
	hipEventCreate(&cuda_parm.t_start);
	hipEventCreate(&cuda_parm.t_stop);
	hipEventRecord(cuda_parm.t_start,cuda_parm.stream[0]);
#endif
	// decide block and thread
	int blocknum=648;
	while(blocknum!=8)
	{
		if(n%blocknum==0 && n/blocknum>=16)
		{
			break;
		}
		blocknum-=4;
	}
	
	dim3 threadnum(8,4);
	size_t s_size = 0;
	int num_per_block = n / blocknum;
	blocknum = blocknum/4;
	dim3 bb(blocknum, codeword_num);
	
	llr_t ext2[n*codeword_num];
	memset(ext2,0,sizeof(ext2));
	llr_t tmp[n*codeword_num];
	
	// for crc check
	char check=0;
	int z;
	// log map algorithm
	log<<<bb, threadnum, s_size, cuda_parm.stream[0]>>>(turbo_parm->sys_d, turbo_parm->sys_d, turbo_parm->sys2_d,
											  turbo_parm->ypar1_d,
											  turbo_parm->alpha_d,
											  turbo_parm->alpha_pre_1, turbo_parm->alpha_pre_2, turbo_parm->beta_pre_1, turbo_parm->beta_pre_2,
											  turbo_parm->ext_d, turbo_parm->ext2_d, turbo_parm->decode_ext2,
											  num_per_block, iteration_cnt, 1, n, codeword_num
											);
	
	while(iteration_cnt++ < max_iterations)
	{
		
		log<<<bb, threadnum, s_size, cuda_parm.stream[0]>>>(turbo_parm->sys_d, turbo_parm->sys1_d, turbo_parm->sys2_d,
												  turbo_parm->ypar2_d,
												  turbo_parm->alpha_d,
												  turbo_parm->alpha_pre_1, turbo_parm->alpha_pre_2, turbo_parm->beta_pre_1, turbo_parm->beta_pre_2,
												  turbo_parm->ext_d, turbo_parm->ext2_d, turbo_parm->decode_ext2,
												  num_per_block, iteration_cnt, 2, n, codeword_num
												);
		
		if(iteration_cnt>=3)
		{
			hipEventRecord(cuda_parm.s_check[iteration_cnt-3], cuda_parm.stream[0]);	
		}

		if(iteration_cnt < max_iterations)
		{
			log<<<bb, threadnum, s_size, cuda_parm.stream[0]>>>(turbo_parm->sys_d, turbo_parm->sys1_d, turbo_parm->sys2_d,
													  turbo_parm->ypar1_d,
													  turbo_parm->alpha_d,
													  turbo_parm->alpha_pre_1, turbo_parm->alpha_pre_2, turbo_parm->beta_pre_1, turbo_parm->beta_pre_2,
													  turbo_parm->ext_d, turbo_parm->ext2_d, turbo_parm->decode_ext2,
													  num_per_block, iteration_cnt, 1, n, codeword_num
													);
		}
	}
	hipDeviceSynchronize();
	
#ifdef TIME_EST
	hipEventRecord(cuda_parm.t_stop, cuda_parm.stream[0]);
	hipEventSynchronize(cuda_parm.t_stop);
	hipEventElapsedTime(&cuda_parm.t_time, cuda_parm.t_start, cuda_parm.t_stop);
	
	printf("Time For turbo algorithm Kernel = %f ms\n",cuda_parm.t_time);	
#endif	

	int decode_len = n >> 3;
	for(i=0;i<=2;i++)
	{
		check = 0;
		// wait for turbo ext
		hipStreamWaitEvent(cuda_parm.stream[1], cuda_parm.s_check[i], 0);
#ifdef TIME_EST
	cuda_parm.d_time = 0;
	hipEventCreate(&cuda_parm.d_start);
	hipEventCreate(&cuda_parm.d_stop);
	hipEventRecord(cuda_parm.d_start, cuda_parm.stream[1]);
#endif
		// decode
		decode<<<codeword_num, 256, 0, cuda_parm.stream[1]>>>(turbo_parm->decode_ext2, turbo_parm->decode_d, turbo_parm->decode_tmp, n, decode_len, i);
		hipStreamSynchronize(cuda_parm.stream[1]);

		// crc check
		for(j=0;j<codeword_num;j++)
		{
			F = f_tmp[1];
			if(j==0)
			{
				F = f_tmp[0];
			}
			oldcrc = *((unsigned int *)(&turbo_parm->decode_h[i*codeword_num*decode_len + j*decode_len + decode_len-crc_len]));
			switch(crc_type)
			{
				case CRC24_A:
					oldcrc&=0x00ffffff;
					crc = crc24a(&turbo_parm->decode_h[i*codeword_num*decode_len + j*decode_len + ( F>>3 )], n-24-F)>>8;
					temp=((uint8_t *)&crc)[2];
					((uint8_t *)&crc)[2] = ((uint8_t *)&crc)[0];
					((uint8_t *)&crc)[0] = temp;
					break;
				case CRC24_B:
					oldcrc&=0x00ffffff;
					crc = crc24b(&turbo_parm->decode_h[i*codeword_num*decode_len + j*decode_len],
								 n-24)>>8;
					temp=((uint8_t *)&crc)[2];
					((uint8_t *)&crc)[2] = ((uint8_t *)&crc)[0];
					((uint8_t *)&crc)[0] = temp;
					break;

				case CRC16:
					oldcrc&=0x0000ffff;
					crc = crc16(&turbo_parm->decode_h[i*codeword_num*decode_len + j*decode_len],
								n-16)>>16;
					break;

				case CRC8:
					oldcrc&=0x000000ff;
					crc = crc8(&turbo_parm->decode_h[i*codeword_num*decode_len + j*decode_len],
							   n-8)>>24;
					break;

				default:
					printf("FATAL: 3gpplte_turbo_decoder_sse.c: Unknown CRC\n");
					return(255);
					break;
			}

			if ((crc == oldcrc) && (crc!=0)) {
				ret[j] = i+3-1;
			}
			else
			{
				check = 1;
			}			
		}

#ifdef TIME_EST
	hipEventRecord(cuda_parm.d_stop, cuda_parm.stream[1]);
	hipEventSynchronize(cuda_parm.d_stop);
	hipEventElapsedTime(&cuda_parm.d_time, cuda_parm.d_start, cuda_parm.d_stop);
	printf("Time For decode & crc check = %f ms\n",cuda_parm.d_time);
#endif	
		if(check==0)
		{
			for(j=0;j<codeword_num;j++)
			{
				for(k=0;k<decode_len;k++)
				{
					decoded_bytes[j][k] = turbo_parm->decode_h[i*codeword_num*decode_len + j*decode_len + k];
				}
			}
			
#ifdef TIME_EST
	hipEventRecord(cuda_parm.e_stop, cuda_parm.stream[0]);
	hipEventSynchronize(cuda_parm.e_stop);
	hipEventElapsedTime(&cuda_parm.e_time, cuda_parm.e_start, cuda_parm.e_stop);
	printf("Time For CUDA = %f ms\n",cuda_parm.e_time);
#endif
			return i+3-1;
		}
	}

	// crc check fail
	for(i=0;i<codeword_num;i++)
	{
		for(j=0;j<decode_len;j++)
		{
			decoded_bytes[i][j] = turbo_parm->decode_h[2*codeword_num*decode_len + i*decode_len + j];
		}
		ret[i] = 5;
		//return 5;
	}
	
#ifdef TIME_EST
	hipEventRecord(cuda_parm.e_stop, cuda_parm.stream[0]);
	hipEventSynchronize(cuda_parm.e_stop);
	hipEventElapsedTime(&cuda_parm.e_time, cuda_parm.e_start, cuda_parm.e_stop);
	printf("Time For CUDA = %f ms\n",cuda_parm.e_time);
#endif	
	for(i=0;i<2;i++)
	{
		hipStreamSynchronize(cuda_parm.stream[i]);
	}
    return 5;
}